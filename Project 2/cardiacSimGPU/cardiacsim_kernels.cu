#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>

using namespace std;

// Kernels

__global__ void update_domain_boundaries(double *E_prev, size_t num_rows, size_t num_cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	E_prev[j][0] = E_prev[j][2];
	E_prev[j][n + 1] = E_prev[j][n - 1];
	E_prev[0][i] = E_prev[2][i];
	E_prev[m + 1][i] = E_prev[m - 1][i];
}

// Version 1 kernels

__global__ void solve_pde_excitation(double *E, double *E_prev, const double alpha)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E_prev[j][i] + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * E_prev[j][i] + E_prev[j + 1][i] + E_prev[j - 1][i]);
}

__global__ void solve_ode_excitation(double *E, double *E_prev, double *R,
									 const double kk, const double dt, const double a)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E[j][i] - dt * (kk * E[j][i] * (E[j][i] - a) * (E[j][i] - 1) + E[j][i] * R[j][i]);
}

__global__ void solve_ode_recovery(double *E, double *R, const double kk, const double dt,
								   const double epsilon, const double M1, const double M2, const double b)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	R[j][i] = R[j][i] + dt * (epsilon + M1 * R[j][i] / (E[j][i] + M2)) * (-R[j][i] - kk * E[j][i] * (E[j][i] - b - 1));
}

// Version 2 kernel

__global__ void simulate_kernel_v2(double *E, double *E_prev, double *R,
						const double alpha, const int n, const int m, const double kk,
						const double dt, const double a, const double epsilon,
						const double M1, const double M2, const double b)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E_prev[j][i] + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * E_prev[j][i] + E_prev[j + 1][i] + E_prev[j - 1][i]);
	E[j][i] = E[j][i] - dt * (kk * E[j][i] * (E[j][i] - a) * (E[j][i] - 1) + E[j][i] * R[j][i]);
	R[j][i] = R[j][i] + dt * (epsilon + M1 * R[j][i] / (E[j][i] + M2)) * (-R[j][i] - kk * E[j][i] * (E[j][i] - b - 1));
}

// Version 3 kernel

__global__ void simulate_kernel_v3(double *E, double *E_prev, double *R,
						const double alpha, const int n, const int m, const double kk,
						const double dt, const double a, const double epsilon,
						const double M1, const double M2, const double b)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int e_ji = E[j][i], e_prev_ji = E_prev[j][i], r_ji = R[j][i];

	E[j][i] = e_prev_ji + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * e_prev_ji + E_prev[j + 1][i] + E_prev[j - 1][i]);
	E[j][i] = e_ji - dt * (kk * e_ji * (e_ji - a) * (e_ji - 1) + e_ji * r_ji);
	R[j][i] = r_ji + dt * (epsilon + M1 * r_ji / (e_ji + M2)) * (-r_ji - kk * e_ji * (e_ji - b - 1));
}

extern "C" void splot(double **E, double T, int niter, int m, int n);
void cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &kernel_no);

static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if (RC == -1)
	{
		cerr << "ERROR: Bad call to gettimeofday" << endl;
		return (-1);
	}

	return (((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec));

}

double **alloc2D(int num_rows, int num_cols)
{
	double **E;
	int nx = num_cols, ny = num_rows;

	E = (double **)malloc(sizeof(double *) * ny + sizeof(double) * nx * ny);
	assert(E);
	int j;
	for (j = 0; j < ny; j++)
		E[j] = (double *)(E + ny) + j * nx;
	return (E);
}

double stats(double **E, int num_rows, int num_cols, double *_mx)
{
	double mx = -1;
	double l2norm = 0;
	int i, j;

	for (i = 1; i <= num_rows; i++)
		for (j = 1; j <= num_cols; j++)
		{
			l2norm += E[i][j] * E[i][j];
			if (E[i][j] > mx)
				mx = E[i][j];
		}

	*_mx = mx;
	l2norm /= (double)((num_rows) * (num_cols));
	l2norm = sqrt(l2norm);
	return l2norm;
}

int main(int argc, char **argv)
{
	double **h_E, **h_R, **h_E_prev;
	double *d_E, *d_R, *d_E_prev;

	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int m = 200, n = 200;
	int plot_freq = 0;
	int bx = 1, by = 1;
	int kernel = 1;

	cmdLine(argc, argv, T, n, bx, by, plot_freq, kernel);

	m = n;

	const dim3 block_size(bx, by);
	const dim3 grid(n / block_size.x, m / block_size.y);

	h_E = alloc2D(m + 2, n + 2);
	h_E_prev = alloc2D(m + 2, n + 2);
	h_R = alloc2D(m + 2, n + 2);

	int i, j;

	for (j = 1; j <= m; j++)
		for (i = 1; i <= n; i++)
			h_E_prev[j][i] = h_R[j][i] = 0;

	for (j = 1; j <= m; j++)
		for (i = n / 2 + 1; i <= n; i++)
			h_E_prev[j][i] = 1.0;

	for (j = m / 2 + 1; j <= m; j++)
		for (i = 1; i <= n; i++)
			h_R[j][i] = 1.0;

	hipMalloc(&d_E, sizeof(double) * (m + 2) * (n + 2));
	hipMalloc(&d_E_prev, sizeof(double) * (m + 2) * (n + 2));
	hipMalloc(&d_R, sizeof(double) * (m + 2) * (n + 2));

	hipMemcpy(d_E, h_E[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, h_E_prev[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_R, h_R[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);

	double dx = 1.0 / n;

	double rp = kk * (b + 1) * (b + 1) / 4;
	double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
	double dtr = 1 / (epsilon + ((M1 / M2) * rp));
	double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
	double alpha = d * dt / (dx * dx);

	cout << "Grid Size       : " << n << endl;
	cout << "Duration of Sim : " << T << endl;
	cout << "Time step dt    : " << dt << endl;
	cout << "Block Size: " << bx << " x " << by << endl;
	cout << "Using CUDA Kernel Version: " << kernel << endl;

	cout << endl;

	double t0 = getTime();

	double t = 0.0;
	int niter = 0;

	while (t < T)
	{
		t += dt;
		niter++;

		update_domain_boundaries<<<grid, block_size>>>(d_E_prev, m, n);

		hipDeviceSynchronize();

		if (kernel == 1)
		{
			solve_pde_excitation<<<grid, block_size>>>(d_E, d_E_prev, alpha);

			hipDeviceSynchronize();

			solve_ode_excitation<<<grid, block_size>>>(d_E, d_E_prev, d_R, kk, dt, a);

			hipDeviceSynchronize();

			solve_ode_recovery<<<grid, block_size>>>(d_E, d_R, kk, dt, epsilon, M1, M2, b);
		}
		else if (kernel == 2)
		{
			simulate_kernel_v2<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		} 
		else if (kernel == 3) {
			simulate_kernel_v3<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		}

		hipDeviceSynchronize();

		double *tmp = d_E;
		d_E = d_E_prev;
		d_E_prev = tmp;

		// hipMemcpy(&h_E[0], d_E, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);

		// if (plot_freq)
		// {
		// 	int k = (int)(t / plot_freq);
		// 	if ((t - k * plot_freq) < dt)
		// 	{
		// 		splot(h_E, t, niter, m + 2, n + 2);
		// 	}
		// }
	} //end of while loop

	double time_elapsed = getTime() - t0;

	double Gflops = (double)(niter * (1E-9 * n * n) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;

	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl;
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl
		 << endl;

	hipMemcpy(h_E_prev[0], d_E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);
	hipMemcpy(h_R[0], d_R, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);

	double mx;
	double l2norm = stats(h_E_prev, m, n, &mx);
	cout << "Max: " << mx << " L2norm: " << l2norm << endl;

	if (plot_freq)
	{
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

	free(h_E);
	free(h_E_prev);
	free(h_R);

	hipFree(d_E);
	hipFree(d_E_prev);
	hipFree(d_R);

	return 0;
}

void cmdLine(int argc, char *argv[], double &T, int &n, int &bx, int &by, int &plot_freq, int &kernel)
{
	static struct option long_options[] = {
		{"n", required_argument, 0, 'n'},
		{"bx", required_argument, 0, 'x'},
		{"by", required_argument, 0, 'y'},
		{"tfinal", required_argument, 0, 't'},
		{"plot", required_argument, 0, 'p'},
		{"kernel_version", required_argument, 0, 'v'},
	};

	int ac;
	for (ac = 1; ac < argc; ac++)
	{
		int c;
		while ((c = getopt_long(argc, argv, "n:x:y:t:p:v:", long_options, NULL)) != -1)
		{
			switch (c)
			{

				// Size of the computational box
			case 'n':
				n = atoi(optarg);
				break;

				// X block geometry
			case 'x':
				bx = atoi(optarg);

				// Y block geometry
			case 'y':
				by = atoi(optarg);

				// Length of simulation, in simulated time units
			case 't':
				T = atof(optarg);
				break;

				// Plot the excitation variable
			case 'p':
				plot_freq = atoi(optarg);
				break;

				// Kernel version
			case 'v':
				kernel = atoi(optarg);
				break;

				// Error
			default:
				printf("Usage:  [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-x <x block geometry> [-y <y block geometry][-v <Kernel Version>]\n");
				exit(-1);
			}
		}
	}
}

FILE *gnu = NULL;

void splot(double **U, double T, int niter, int m, int n)
{
	int i, j;
	if (gnu == NULL)
		gnu = popen("gnuplot", "w");

	double mx = -1, mn = 32768;
	for (j = 0; j < m; j++)
		for (i = 0; i < n; i++)
		{
			if (U[j][i] > mx)
				mx = U[j][i];
			if (U[j][i] < mn)
				mn = U[j][i];
		}

	fprintf(gnu, "set title \"T = %f [niter = %d]\"\n", T, niter);
	fprintf(gnu, "set size square\n");
	fprintf(gnu, "set key off\n");
	fprintf(gnu, "set pm3d map\n");
	fprintf(gnu, "set palette defined (-3 \"blue\", 0 \"white\", 1 \"red\")\n");
	fprintf(gnu, "splot [0:%d] [0:%d][%f:%f] \"-\"\n", m - 1, n - 1, mn, mx);
	
	for (j = 0; j < m; j++)
	{
		for (i = 0; i < n; i++)
		{
			fprintf(gnu, "%d %d %f\n", i, j, U[i][j]);
		}
		fprintf(gnu, "\n");
	}
	fprintf(gnu, "e\n");
	fflush(gnu);
	return;
}
