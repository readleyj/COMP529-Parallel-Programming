/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 * Refer to "Detailed Numerical Analyses of the Aliev-Panfilov Model on GPGPU"
 * https://www.simula.no/publications/detailed-numerical-analyses-aliev-panfilov-model-gpgpu
 * by Xing Cai, Didem Unat and Scott Baden
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>

using namespace std;

// Kernels

__global__ void update_domain_boundaries(double **E_prev, size_t m, size_t n)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E_prev[j][0] = E_prev[j][2];
	E_prev[j][n + 1] = E_prev[j][n - 1];
	E_prev[0][i] = E_prev[2][i];
	E_prev[m + 1][i] = E_prev[m - 1][i];
}

// Version 1 kernels

__global__ void solve_pde_excitation(double **E, double **E_prev, const double alpha)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E_prev[j][i] + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * E_prev[j][i] + E_prev[j + 1][i] + E_prev[j - 1][i]);
}

__global__ void solve_ode_excitation(double **E, double **E_prev, double **R,
									 const double kk, const double dt, const double a)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E[j][i] - dt * (kk * E[j][i] * (E[j][i] - a) * (E[j][i] - 1) + E[j][i] * R[j][i]);
}

__global__ void solve_ode_recovery(double **E, double **R, const double kk, const double dt,
								   const double epsilon, const double M1, const double M2, const double b)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	R[j][i] = R[j][i] + dt * (epsilon + M1 * R[j][i] / (E[j][i] + M2)) * (-R[j][i] - kk * E[j][i] * (E[j][i] - b - 1));
}

// Version 2 kernel

__global__ void simulate_kernel_v2(double **E, double **E_prev, double **R,
						const double alpha, const int n, const int m, const double kk,
						const double dt, const double a, const double epsilon,
						const double M1, const double M2, const double b)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	E[j][i] = E_prev[j][i] + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * E_prev[j][i] + E_prev[j + 1][i] + E_prev[j - 1][i]);
	E[j][i] = E[j][i] - dt * (kk * E[j][i] * (E[j][i] - a) * (E[j][i] - 1) + E[j][i] * R[j][i]);
	R[j][i] = R[j][i] + dt * (epsilon + M1 * R[j][i] / (E[j][i] + M2)) * (-R[j][i] - kk * E[j][i] * (E[j][i] - b - 1));
}

// Version 3 kernel

__global__ void simulate_kernel_v3(double **E, double **E_prev, double **R,
						const double alpha, const int n, const int m, const double kk,
						const double dt, const double a, const double epsilon,
						const double M1, const double M2, const double b)
{
	printf("Hello\n");
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int e_ji = E[j][i], e_prev_ji = E_prev[j][i], r_ji = R[j][i];

	E[j][i] = e_prev_ji + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * e_prev_ji + E_prev[j + 1][i] + E_prev[j - 1][i]);
	E[j][i] = e_ji - dt * (kk * e_ji * (e_ji - a) * (e_ji - 1) + e_ji * r_ji);
	R[j][i] = r_ji + dt * (epsilon + M1 * r_ji / (e_ji + M2)) * (-r_ji - kk * e_ji * (e_ji - b - 1));
}

// External functions
extern "C" void splot(double **E, double T, int niter, int m, int n);

void cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &kernel_no);

// Utilities
//

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if (RC == -1)
	{
		cerr << "ERROR: Bad call to gettimeofday" << endl;
		return (-1);
	}

	return (((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec));

} // end getTime()

// Allocate a 2D array
double **alloc2D(int m, int n)
{
	double **E;
	int nx = n, ny = m;
	E = (double **)malloc(sizeof(double *) * ny + sizeof(double) * nx * ny);
	assert(E);
	int j;
	for (j = 0; j < ny; j++)
		E[j] = (double *)(E + ny) + j * nx;
	return (E);
}

// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double **E, int m, int n, double *_mx)
{
	double mx = -1;
	double l2norm = 0;
	int i, j;
	for (j = 1; j <= m; j++)
		for (i = 1; i <= n; i++)
		{
			l2norm += E[j][i] * E[j][i];
			if (E[j][i] > mx)
				mx = E[j][i];
		}
	*_mx = mx;
	l2norm /= (double)((m) * (n));
	l2norm = sqrt(l2norm);
	return l2norm;
}

void simulate(double **E, double **E_prev, double **R,
			  const double alpha, const int n, const int m, const double kk,
			  const double dt, const double a, const double epsilon,
			  const double M1, const double M2, const double b)
{
	int i, j;
	/* 
     * Copy data from boundary of the computational box 
     * to the padding region, set up for differencing
     * on the boundary of the computational box
     * Using mirror boundaries
     */

	for (j = 1; j <= m; j++)
		E_prev[j][0] = E_prev[j][2];
	for (j = 1; j <= m; j++)
		E_prev[j][n + 1] = E_prev[j][n - 1];

	for (i = 1; i <= n; i++)
		E_prev[0][i] = E_prev[2][i];
	for (i = 1; i <= n; i++)
		E_prev[m + 1][i] = E_prev[m - 1][i];

	// Solve for the excitation, the PDE
	for (j = 1; j <= m; j++)
	{
		for (i = 1; i <= n; i++)
		{
			E[j][i] = E_prev[j][i] + alpha * (E_prev[j][i + 1] + E_prev[j][i - 1] - 4 * E_prev[j][i] + E_prev[j + 1][i] + E_prev[j - 1][i]);
		}
	}

	/* 
     * Solve the ODE, advancing excitation and recovery to the
     *     next timtestep
     */
	for (j = 1; j <= m; j++)
	{
		for (i = 1; i <= n; i++)
			E[j][i] = E[j][i] - dt * (kk * E[j][i] * (E[j][i] - a) * (E[j][i] - 1) + E[j][i] * R[j][i]);
	}

	for (j = 1; j <= m; j++)
	{
		for (i = 1; i <= n; i++)
			R[j][i] = R[j][i] + dt * (epsilon + M1 * R[j][i] / (E[j][i] + M2)) * (-R[j][i] - kk * E[j][i] * (E[j][i] - b - 1));
	}
}

// Main program
int main(int argc, char **argv)
{
	/*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
	double **h_E, **h_R, **h_E_prev;
	double **d_E, **d_R, **d_E_prev;

	// Various constants - these definitions shouldn't change
	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int m = 200, n = 200;
	int plot_freq = 0;
	int bx = 1, by = 1;
	int kernel = 1;

	cmdLine(argc, argv, T, n, bx, by, plot_freq, kernel);

	m = n;

	const dim3 block_size(bx, by);
	const dim3 grid(n / block_size.x, m / block_size.y);

	// Allocate contiguous memory for solution arrays
	// The computational box is defined on [1:m+1,1:n+1]
	// We pad the arrays in order to facilitate differencing on the
	// boundaries of the computation box
	h_E = alloc2D(m + 2, n + 2);
	h_E_prev = alloc2D(m + 2, n + 2);
	h_R = alloc2D(m + 2, n + 2);

	int i, j;
	// Initialization
	for (j = 1; j <= m; j++)
		for (i = 1; i <= n; i++)
			h_E_prev[j][i] = h_R[j][i] = 0;

	for (j = 1; j <= m; j++)
		for (i = n / 2 + 1; i <= n; i++)
			h_E_prev[j][i] = 1.0;

	for (j = m / 2 + 1; j <= m; j++)
		for (i = 1; i <= n; i++)
			h_R[j][i] = 1.0;

	hipMalloc((void **)&d_E, sizeof(double) * (m + 2) * (n + 2));
	hipMalloc((void **)&d_E_prev, sizeof(double) * (m + 2) * (n + 2));
	hipMalloc((void **)&d_R, sizeof(double) * (m + 2) * (n + 2));

	hipMemcpy(d_E, &h_E[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, &h_E_prev[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_R, &h_R[0], sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);

	double dx = 1.0 / n;

	// For time integration, these values shouldn't change
	double rp = kk * (b + 1) * (b + 1) / 4;
	double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
	double dtr = 1 / (epsilon + ((M1 / M2) * rp));
	double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
	double alpha = d * dt / (dx * dx);

	cout << "Grid Size       : " << n << endl;
	cout << "Duration of Sim : " << T << endl;
	cout << "Time step dt    : " << dt << endl;
	cout << "Block Size: " << bx << " x " << by << endl;
	cout << "Using CUDA Kernel Version: " << kernel << endl;

	cout << endl;

	// Start the timer
	double t0 = getTime();

	// Simulated time is different from the integer timestep number
	// Simulated time
	double t = 0.0;
	// Integer timestep number
	int niter = 0;

	while (t < T)
	{
		t += dt;
		niter++;

		update_domain_boundaries<<<grid, block_size>>>(d_E_prev, m, n);

		hipDeviceSynchronize();

		if (kernel == 1)
		{
			solve_pde_excitation<<<grid, block_size>>>(d_E, d_E_prev, alpha);

			hipDeviceSynchronize();

			solve_ode_excitation<<<grid, block_size>>>(d_E, d_E_prev, d_R, kk, dt, a);

			hipDeviceSynchronize();

			solve_ode_recovery<<<grid, block_size>>>(d_E, d_R, kk, dt, epsilon, M1, M2, b);
		}
		else if (kernel == 2)
		{
			simulate_kernel_v2<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		} 
		else if (kernel == 3) {
			simulate_kernel_v3<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
		}

		hipDeviceSynchronize();

		//swap current E with previous E
		double **tmp = d_E;
		d_E = d_E_prev;
		d_E_prev = tmp;

		// cudaMemcpy(&h_E[0], d_E, sizeof(double) * (m + 2) * (n + 2), cudaMemcpyDeviceToHost);

		// if (plot_freq)
		// {
		// 	int k = (int)(t / plot_freq);
		// 	if ((t - k * plot_freq) < dt)
		// 	{
		// 		splot(h_E, t, niter, m + 2, n + 2);
		// 	}
		// }
	} //end of while loop

	double time_elapsed = getTime() - t0;

	double Gflops = (double)(niter * (1E-9 * n * n) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;

	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl;
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl
		 << endl;

	hipMemcpy(&h_E_prev[0], d_E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);
	hipMemcpy(&h_R[0], d_R, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);

	double mx;
	double l2norm = stats(h_E_prev, m, n, &mx);
	cout << "Max: " << mx << " L2norm: " << l2norm << endl;

	if (plot_freq)
	{
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

	free(h_E);
	free(h_E_prev);
	free(h_R);

	hipFree(d_E);
	hipFree(d_E_prev);
	hipFree(d_R);

	return 0;
}

void cmdLine(int argc, char *argv[], double &T, int &n, int &bx, int &by, int &plot_freq, int &kernel)
{
	/// Command line arguments
	// Default value of the domain sizes
	static struct option long_options[] = {
		{"n", required_argument, 0, 'n'},
		{"bx", required_argument, 0, 'x'},
		{"by", required_argument, 0, 'y'},
		{"tfinal", required_argument, 0, 't'},
		{"plot", required_argument, 0, 'p'},
		{"kernel_version", required_argument, 0, 'v'},
	};
	// Process command line arguments
	int ac;
	for (ac = 1; ac < argc; ac++)
	{
		int c;
		while ((c = getopt_long(argc, argv, "n:x:y:t:p:v:", long_options, NULL)) != -1)
		{
			switch (c)
			{

				// Size of the computational box
			case 'n':
				n = atoi(optarg);
				break;

				// X block geometry
			case 'x':
				bx = atoi(optarg);

				// Y block geometry
			case 'y':
				by = atoi(optarg);

				// Length of simulation, in simulated time units
			case 't':
				T = atof(optarg);
				break;

				// Plot the excitation variable
			case 'p':
				plot_freq = atoi(optarg);
				break;

				// Kernel version
			case 'v':
				kernel = atoi(optarg);
				break;

				// Error
			default:
				printf("Usage:  [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-x <x block geometry> [-y <y block geometry][-v <Kernel Version>]\n");
				exit(-1);
			}
		}
	}
}
/* **********************************************************
 *  Author : Urvashi R.V. [04/06/2004]
 *      Modified by Didem Unat [03/23/21]
 *************************************************************/

#include <stdio.h>

/* Function to plot the 2D array
 * 'gnuplot' is instantiated via a pipe and 
 * the values to be plotted are passed through, along 
 * with gnuplot commands */

FILE *gnu = NULL;

void splot(double **U, double T, int niter, int m, int n)
{
	int i, j;
	if (gnu == NULL)
		gnu = popen("gnuplot", "w");

	double mx = -1, mn = 32768;
	for (j = 0; j < m; j++)
		for (i = 0; i < n; i++)
		{
			if (U[j][i] > mx)
				mx = U[j][i];
			if (U[j][i] < mn)
				mn = U[j][i];
		}

	fprintf(gnu, "set title \"T = %f [niter = %d]\"\n", T, niter);
	fprintf(gnu, "set size square\n");
	fprintf(gnu, "set key off\n");
	fprintf(gnu, "set pm3d map\n");
	// Various color schemes
	fprintf(gnu, "set palette defined (-3 \"blue\", 0 \"white\", 1 \"red\")\n");

	//    fprintf(gnu,"set palette rgbformulae 22, 13, 31\n");
	//    fprintf(gnu,"set palette rgbformulae 30, 31, 32\n");

	fprintf(gnu, "splot [0:%d] [0:%d][%f:%f] \"-\"\n", m - 1, n - 1, mn, mx);
	for (j = 0; j < m; j++)
	{
		for (i = 0; i < n; i++)
		{
			fprintf(gnu, "%d %d %f\n", i, j, U[i][j]);
		}
		fprintf(gnu, "\n");
	}
	fprintf(gnu, "e\n");
	fflush(gnu);
	return;
}
