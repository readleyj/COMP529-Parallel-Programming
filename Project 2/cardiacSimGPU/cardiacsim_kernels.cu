
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>

using namespace std;

#define RADIUS 1

// Kernels

__global__ void update_domain_boundaries(double *E_prev, size_t height, size_t width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		E_prev[(row * width) + 0] = E_prev[(row * width) + 2];
		E_prev[(row * width) + width + 1] = E_prev[(row * width) + width - 1];
		E_prev[(0 * width) + col] = E_prev[(2 * width) + col];
		E_prev[(height * (width + 1)) + col] = E_prev[(height * (width - 1)) + col];
	}
}

// Version 1 kernels

__global__ void solve_pde_excitation(double *E, double *E_prev, const double alpha,
									 size_t height, size_t width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int idx_in = row * width + col;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		E[idx_in] = E_prev[idx_in] + alpha * (E_prev[idx_in + 1] + E_prev[idx_in + 1] - 4 * E_prev[idx_in] + E_prev[idx_in + width] + E_prev[idx_in - width]) + 10;
	}
}

__global__ void solve_ode_excitation(double *E, double *E_prev, double *R,
									 const double kk, const double dt, const double a,
									 size_t height, size_t width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int idx_in = row * width + col;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		E[idx_in] = E[idx_in] - dt * (kk * E[idx_in] * (E[idx_in] - a) * (E[idx_in] - 1) + E[idx_in] * R[idx_in]) + 10;
	}
}

__global__ void solve_ode_recovery(double *E, double *R, const double kk, const double dt,
								   const double epsilon, const double M1, const double M2,
								   const double b, size_t height, size_t width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int idx_in = row * width + col;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		R[idx_in] = R[idx_in] + dt * (epsilon + M1 * R[idx_in] / (E[idx_in] + M2)) *
									(-R[idx_in] - kk * E[idx_in] * (E[idx_in] - b - 1));
	}
}

// Version 2 kernel

__global__ void simulate_kernel_v2(double *E, double *E_prev, double *R,
								   const double alpha, const double kk,
								   const double dt, const double a, const double epsilon,
								   const double M1, const double M2, const double b,
								   size_t height, size_t width)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int idx_in = row + col * width;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		E[idx_in] = E_prev[idx_in] + alpha * (E_prev[idx_in + col * width] +
											  E_prev[idx_in - col * width] -
											  4 * E_prev[idx_in] +
											  E_prev[idx_in + 1] +
											  E_prev[idx_in - 1]);

		E[idx_in] = E[idx_in] - dt * (kk * E[idx_in] * (E[idx_in] - a) * (E[idx_in] - 1) + E[idx_in] * R[idx_in]);

		R[idx_in] = R[idx_in] + dt * (epsilon + M1 * R[idx_in] / (E[row + col * width] + M2)) *
									(-R[idx_in] - kk * E[idx_in] * (E[idx_in] - b - 1));
	}
}

// Version 3 kernel

__global__ void simulate_kernel_v3(double *E, double *E_prev, double *R,
								   const double alpha, const double kk,
								   const double dt, const double a, const double epsilon,
								   const double M1, const double M2, const double b,
								   size_t height, size_t width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int idx_in = row + col * width;

	if (row >= RADIUS && col >= RADIUS && row < height - RADIUS && col < width - RADIUS)
	{
		int e_ji = E[idx_in], e_prev_ji = E_prev[idx_in], r_ji = R[idx_in];

		E[idx_in] = e_prev_ji + alpha * (E_prev[idx_in + col * width] + E_prev[idx_in - col * width] - 4 * e_prev_ji + E_prev[idx_in + 1] + E_prev[idx_in - 1]);
		E[idx_in] = e_ji - dt * (kk * e_ji * (e_ji - a) * (e_ji - 1) + e_ji * r_ji);
		R[idx_in] = r_ji + dt * (epsilon + M1 * r_ji / (e_ji + M2)) * (-r_ji - kk * e_ji * (e_ji - b - 1));
	}
}

extern "C" void splot(double **E, double T, int niter, int m, int n);
void cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &kernel_no);

static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if (RC == -1)
	{
		cerr << "ERROR: Bad call to gettimeofday" << endl;
		return (-1);
	}

	return (((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec));
}

double **alloc2D(int height, int width)
{
	double **E;
	int nx = width, ny = height;

	E = (double **)malloc(sizeof(double *) * ny + sizeof(double) * nx * ny);
	assert(E);
	int j;
	for (j = 0; j < ny; j++)
		E[j] = (double *)(E + ny) + j * nx;
	return (E);
}

double stats(double **E, int height, int width, double *_mx)
{
	double mx = -1;
	double l2norm = 0;
	int i, j;

	for (i = 1; i <= height; i++)
		for (j = 1; j <= width; j++)
		{
			l2norm += E[i][j] * E[i][j];
			if (E[i][j] > mx)
				mx = E[i][j];
		}

	*_mx = mx;
	l2norm /= (double)((height) * (width));
	l2norm = sqrt(l2norm);
	return l2norm;
}

int main(int argc, char **argv)
{
	double **h_E, **h_R, **h_E_prev;
	double *d_E, *d_R, *d_E_prev;

	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int m = 200, n = 200;
	int plot_freq = 0;
	int bx = 1, by = 1;
	int kernel = 1;

	cmdLine(argc, argv, T, n, bx, by, plot_freq, kernel);

	m = n;

	int height = m + 2, width = n + 2;

	const dim3 block_size(bx, by);
	const dim3 grid(n / block_size.x, m / block_size.y);

	h_E = alloc2D(height, width);
	h_E_prev = alloc2D(height, width);
	h_R = alloc2D(height, width);

	int i, j;

	for (i = 1; i <= m; i++)
		for (j = 1; j <= n; j++)
			h_E_prev[i][j] = h_R[i][j] = 0;

	for (i = 1; i <= m; i++)
		for (j = n / 2 + 1; j <= n; j++)
			h_E_prev[i][j] = 1.0;

	for (i = m / 2 + 1; i <= m; i++)
		for (j = 1; j <= n; j++)
			h_R[i][j] = 1.0;

	hipMalloc(&d_E, sizeof(double) * height * width);
	hipMalloc(&d_E_prev, sizeof(double) * height * width);
	hipMalloc(&d_R, sizeof(double) * height * width);

	hipMemcpy(d_E, h_E[0], sizeof(double) * height * width, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, h_E_prev[0], sizeof(double) * height * width, hipMemcpyHostToDevice);
	hipMemcpy(d_R, h_R[0], sizeof(double) * height * width, hipMemcpyHostToDevice);

	double dx = 1.0 / n;

	double rp = kk * (b + 1) * (b + 1) / 4;
	double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
	double dtr = 1 / (epsilon + ((M1 / M2) * rp));
	double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
	double alpha = d * dt / (dx * dx);

	cout << "Grid Size       : " << n << endl;
	cout << "Duration of Sim : " << T << endl;
	cout << "Time step dt    : " << dt << endl;
	cout << "Block Size: " << bx << " x " << by << endl;
	cout << "Using CUDA Kernel Version: " << kernel << endl;

	cout << endl;

	double t0 = getTime();

	double t = 0.0;
	int niter = 0;

	while (t < T)
	{
		t += dt;
		niter++;

		update_domain_boundaries<<<grid, block_size>>>(d_E_prev, m, n);

		hipDeviceSynchronize();

		if (kernel == 1)
		{
			solve_pde_excitation<<<grid, block_size>>>(d_E, d_E_prev, alpha, m, n);

			hipDeviceSynchronize();

			solve_ode_excitation<<<grid, block_size>>>(d_E, d_E_prev, d_R, kk, dt, a, m, n);

			hipDeviceSynchronize();

			solve_ode_recovery<<<grid, block_size>>>(d_E, d_R, kk, dt, epsilon, M1, M2, b, m, n);
		}
		else if (kernel == 2)
		{
			simulate_kernel_v2<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, kk, dt,
													 a, epsilon, M1, M2, b, m, n);
		}
		else if (kernel == 3)
		{
			simulate_kernel_v3<<<grid, block_size>>>(d_E, d_E_prev, d_R, alpha, kk, dt,
													 a, epsilon, M1, M2, b, m, n);
		}

		hipDeviceSynchronize();

		double *tmp = d_E;
		d_E = d_E_prev;
		d_E_prev = tmp;

		if (plot_freq)
		{
			hipMemcpy(h_E[0], d_E, sizeof(double) * height * width, hipMemcpyDeviceToHost);

			int k = (int)(t / plot_freq);
			if ((t - k * plot_freq) < dt)
			{
				splot(h_E, t, niter, height, width);
			}
		}
	}

	double time_elapsed = getTime() - t0;

	double Gflops = (double)(niter * (1E-9 * n * n) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;

	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl;
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl
		 << endl;

	hipMemcpy(h_E_prev[0], d_E_prev, sizeof(double) * height * width, hipMemcpyDeviceToHost);
	hipMemcpy(h_R[0], d_R, sizeof(double) * height * width, hipMemcpyDeviceToHost);

	double mx;
	double l2norm = stats(h_E_prev, m, n, &mx);
	cout << "Max: " << mx << " L2norm: " << l2norm << endl;

	if (plot_freq)
	{
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

	free(h_E);
	free(h_E_prev);
	free(h_R);

	hipFree(d_E);
	hipFree(d_E_prev);
	hipFree(d_R);

	return 0;
}

void cmdLine(int argc, char *argv[], double &T, int &n, int &bx, int &by, int &plot_freq, int &kernel)
{
	static struct option long_options[] = {
		{"n", required_argument, 0, 'n'},
		{"bx", required_argument, 0, 'x'},
		{"by", required_argument, 0, 'y'},
		{"tfinal", required_argument, 0, 't'},
		{"plot", required_argument, 0, 'p'},
		{"kernel_version", required_argument, 0, 'v'},
	};

	int ac;
	for (ac = 1; ac < argc; ac++)
	{
		int c;
		while ((c = getopt_long(argc, argv, "n:x:y:t:p:v:", long_options, NULL)) != -1)
		{
			switch (c)
			{

				// Size of the computational box
			case 'n':
				n = atoi(optarg);
				break;

				// X block geometry
			case 'x':
				bx = atoi(optarg);

				// Y block geometry
			case 'y':
				by = atoi(optarg);

				// Length of simulation, in simulated time units
			case 't':
				T = atof(optarg);
				break;

				// Plot the excitation variable
			case 'p':
				plot_freq = atoi(optarg);
				break;

				// Kernel version
			case 'v':
				kernel = atoi(optarg);
				break;

				// Error
			default:
				printf("Usage:  [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-x <x block geometry> [-y <y block geometry][-v <Kernel Version>]\n");
				exit(-1);
			}
		}
	}
}

FILE *gnu = NULL;

void splot(double **U, double T, int niter, int m, int n)
{
	int i, j;
	if (gnu == NULL)
		gnu = popen("gnuplot", "w");

	double mx = -1, mn = 32768;
	for (j = 0; j < m; j++)
		for (i = 0; i < n; i++)
		{
			if (U[j][i] > mx)
				mx = U[j][i];
			if (U[j][i] < mn)
				mn = U[j][i];
		}

	fprintf(gnu, "set title \"T = %f [niter = %d]\"\n", T, niter);
	fprintf(gnu, "set size square\n");
	fprintf(gnu, "set key off\n");
	fprintf(gnu, "set pm3d map\n");
	fprintf(gnu, "set palette defined (-3 \"blue\", 0 \"white\", 1 \"red\")\n");
	fprintf(gnu, "splot [0:%d] [0:%d][%f:%f] \"-\"\n", m - 1, n - 1, mn, mx);

	for (j = 0; j < m; j++)
	{
		for (i = 0; i < n; i++)
		{
			fprintf(gnu, "%d %d %f\n", i, j, U[i][j]);
		}
		fprintf(gnu, "\n");
	}
	fprintf(gnu, "e\n");
	fflush(gnu);
	return;
}
